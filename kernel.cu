#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include ""
#include "cuda_helper.hpp"

#include <iostream>
#include <stdio.h>

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	try {
		// Allocate buffers on GPU
		allocateCudaBuffer(dev_a, size);
		allocateCudaBuffer(dev_b, size);
		allocateCudaBuffer(dev_c, size);
		// Copy input vectors from host memory to GPU buffers.
		hostToDeviceMemcopy(dev_a, a, size);
		hostToDeviceMemcopy(dev_b, b, size);
	}
	catch (const CudaException& exception) {
		std::cout << exception.what() << std::endl;
		return exception.getErrorCode();
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel <<<1, size >>> (dev_c, dev_a, dev_b);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
