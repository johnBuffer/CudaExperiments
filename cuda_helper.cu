#include "cuda_helper.cuh"

void checkCudaStatus(hipError_t status)
{
	if (status != hipSuccess) {
		throw CudaException(status);
	}
}

void cudaSetDeviceExcept(uint32_t deviceID)
{
	checkCudaStatus(hipSetDevice(deviceID));
}

void cudaLastErrorToException()
{
	checkCudaStatus(hipGetLastError());
}

void cudaWaitForDevice()
{
	checkCudaStatus(hipDeviceSynchronize());
}
